#include <hip/hip_runtime.h>
#include <nccl.h>

#include <iostream>

#define NCCL_CHECK(cmd)                                                    \
  do {                                                                     \
    ncclResult_t res = cmd;                                                \
    if (res != ncclSuccess) {                                              \
      std::cerr << "NCCL error: " << ncclGetErrorString(res) << std::endl; \
      exit(EXIT_FAILURE);                                                  \
    }                                                                      \
  } while (0)

#define CUDA_CHECK(call)                                               \
  do {                                                                 \
    hipError_t err = call;                                            \
    if (err != hipSuccess) {                                          \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " \
                << __FILE__ << ":" << __LINE__ << std::endl;           \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

int main() {
  int size = 2;  // GPU数量
  int rank = 0;  // 当前GPU的rank

  ncclComm_t comm;
  ncclUniqueId id;
  ncclGetUniqueId(&id);
  ncclCommInitRank(&comm, size, id, rank);

  const int N = 1024;
  float* send_buff;
  float* recv_buff;

  CUDA_CHECK(hipMalloc(&send_buff, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&recv_buff, N * sizeof(float)));

  float value = rank + 1;
}