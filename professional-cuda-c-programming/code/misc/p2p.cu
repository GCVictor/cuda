#include <hip/hip_runtime.h>

#include <iostream>

int main(int argc, char** argv) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    for (int j = 0; j < deviceCount; j++) {
      if (i == j) continue;

      int canAccessPeer;
      hipDeviceCanAccessPeer(&canAccessPeer, i, j);

      if (canAccessPeer) {
        hipSetDevice(i);
        hipDeviceEnablePeerAccess(j, 0);  // 启用 P2P 访问
        printf("GPU %d can access GPU %d directly\n", i, j);
      } else {
        printf("GPU %d cannot access GPU %d directly\n", i, j);
      }
    }
  }
}