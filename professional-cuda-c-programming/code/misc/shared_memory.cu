#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

void shared_memory_per_block() {
    int device_id = 0;
    int shared_mem_size;

    hipDeviceGetAttribute(&shared_mem_size, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id);
    std::cout << "Shared Memory Size per Block: " << shared_mem_size << " bytes" << std::endl;
}

__global__ void total_shared_memory() {
    extern __shared__ int shared_memory[];
    printf("Shared Memory Size per Block: %lu bytes\n", sizeof(shared_memory));
}

int main() {
    shared_memory_per_block();
    total_shared_memory<<<1, 1, 1024>>>();

    return 0;
}

// Output: 49152 bytes (48KB)