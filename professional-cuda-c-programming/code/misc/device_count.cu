#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

int main(int argc, char **argv) {
  int ngpus;
  hipGetDeviceCount(&ngpus);

  for (int i = 0; i < ngpus; ++i) {
    hipDeviceProp_t devProp;

    hipGetDeviceProperties(&devProp, i);
    printf("Device %d has compute capability %d.%d.\n", i, devProp.major,
           devProp.minor);
  }

  return 0;
}